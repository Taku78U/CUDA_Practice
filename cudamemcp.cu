
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int* a_d, int* b_d, int* c_d){
  *c_d = *a_d + *b_d;
  return;
}

int main(){
  int a = 1, b = 2;
  int *a_d, *b_d, *c_d;
  hipMalloc((void**) &a_d, sizeof(int));
  hipMalloc((void**) &b_d, sizeof(int));
  hipMalloc((void**) &c_d, sizeof(int));

  hipMemcpy(a_d, &a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(b_d, &b, sizeof(int), hipMemcpyHostToDevice);
  int c;

  kernel<<<1, 1>>>(a_d, b_d, c_d);
  hipMemcpy(&c, c_d, sizeof(int), hipMemcpyDeviceToHost);
  hipFree((void**) a_d);
  hipFree((void**) b_d);
  hipFree((void**) c_d);
  printf("%d\n", c);
}
